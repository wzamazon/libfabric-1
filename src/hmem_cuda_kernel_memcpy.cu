
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2020 Amazon.com, Inc. or its affiliates.
 * All rights reserved.
 *
 * This software is available to you under a choice of one of two
 * licenses.  You may choose to be licensed under the terms of the GNU
 * General Public License (GPL) Version 2, available from the file
 * COPYING in the main directory of this source tree, or the
 * BSD license below:
 *
 *     Redistribution and use in source and binary forms, with or
 *     without modification, are permitted provided that the following
 *     conditions are met:
 *
 *      - Redistributions of source code must retain the above
 *        copyright notice, this list of conditions and the following
 *        disclaimer.
 *
 *      - Redistributions in binary form must reproduce the above
 *        copyright notice, this list of conditions and the following
 *        disclaimer in the documentation and/or other materials
 *        provided with the distribution.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS
 * BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN
 * ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */


__global__
void ofi_cudaKernelMemcpy_kernel(char *dst, char *src, size_t n)
{
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
		dst[i] = src[i];
}

extern "C"
void ofi_cudaKernelMemcpy(char *dst, char* src, size_t n, hipStream_t stream)
{
	size_t numthread = 256;
	size_t numblk = (n - 1)/numthread + 1;

	ofi_cudaKernelMemcpy_kernel<<<numblk, numthread, 0, stream>>>(dst, src, n);
}
